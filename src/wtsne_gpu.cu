#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author      : Zhirong Yang
 Copyright   : Copyright by Zhirong Yang. All rights are reserved.
 Description : Weighted t-SNE with stochastic optimization
 ============================================================================
 */

// Modified by John Lees 2021

#include <cfloat>
#include <hipcub/hipcub.hpp>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>

#include "containers.cuh"
#include "cuda_launch.cuh"
#include "uniform_discrete.hpp"
#include "wtsne.hpp"

/****************************
 * Kernels                  *
 ****************************/

// Updates the embedding
template <typename real_t>
KERNEL void wtsneUpdateYKernel(
    uint32_t * rng_state, const discrete_table_ptrs<real_t> node_table,
    const discrete_table_ptrs<real_t> edge_table, volatile real_t *Y, uint64_t *I,
    uint64_t *J, real_t *Eq, real_t *qsum, uint64_t *qcount, uint64_t nn,
    uint64_t ne, real_t *eta, uint64_t nRepuSamp, real_t nsq, real_t *attrCoef,
    int n_workers) {
  // Worker index based on CUDA launch parameters
  int workerIdx = blockIdx.x * blockDim.x + threadIdx.x;
  const real_t one = 1.0; // Used a few times, fp64/fp32
  if (workerIdx < n_workers) {
    // Bring RNG state into local registers
    interleaved<uint32_t> p_rng(rng_state, workerIdx, n_workers);
    rng_state_t<real_t> rng_block = get_rng_state<real_t>(p_rng);

    real_t dY[DIM];
    real_t Yk_read[DIM];
    real_t Yl_read[DIM];
    real_t c = one / ((*Eq) * nsq);

    real_t qsum_local = 0.0;
    uint64_t qcount_local = 0;

    real_t repuCoef = 2 * c / nRepuSamp * nsq;
    for (int r = 0; r < nRepuSamp + 1; r++) {
      uint64_t k, l;
      if (r == 0) {
        uint64_t e = discrete_draw(rng_block, edge_table) % ne;
        k = I[e];
        l = J[e];
      } else {
        k = discrete_draw(rng_block, node_table) % nn;
        l = discrete_draw(rng_block, node_table) % nn;
      }

      if (k != l) {
        uint64_t lk = k * DIM;
        uint64_t ll = l * DIM;
        real_t dist2 = static_cast<real_t>(0.0);
#pragma unroll
        for (int d = 0; d < DIM; d++) {
          // These are read here to avoid multiple workers writing to the same
          // location below
          Yk_read[d] = Y[d + lk];
          Yl_read[d] = Y[d + ll];
          dY[d] = Yk_read[d] - Yl_read[d];
          dist2 += dY[d] * dY[d];
        }
        __threadfence();

        real_t q = one / (1 + dist2);

        real_t g;
        if (r == 0) {
          g = -*attrCoef * q;
        } else {
          g = repuCoef * q * q;
        }

        bool overwrite = false;
#pragma unroll
        for (int d = 0; d < DIM; d++) {
          real_t gain = *eta * g * dY[d];
          // The atomics below basically do
          // Y[d + lk] += gain;
          // Y[d + ll] -= gain;
          // But try again if another worker has written to the same location
          if (atomicAdd((real_t*)Y + d + lk, gain) != Yk_read[d] ||
              atomicAdd((real_t*)Y + d + ll, -gain) != Yl_read[d]) {
            overwrite = true;
          }
        }
        if (!overwrite) {
          qsum_local += q;
          qcount_local++;
        } else {
          // Reset values
#pragma unroll
          for (int d = 0; d < DIM; d++) {
            Y[d + lk] = Yk_read[d];
            Y[d + ll] = Yl_read[d];
          }
          __threadfence();
          r--;
        }
      }
    }
    __syncwarp();

    // Store local state (RNG & counts) back to global
    put_rng_state(rng_block, p_rng);
    qsum[workerIdx] = qsum_local;
    qcount[workerIdx] = qcount_local;
  }
}

/****************************
 * Classes                  *
 ****************************/
template <typename real_t> struct kernel_ptrs {
  uint32_t *rng;
  real_t *Y;
  uint64_t *I;
  uint64_t *J;
  real_t *Eq;
  real_t *qsum;
  uint64_t *qcount;
  uint64_t nn;
  uint64_t ne;
  real_t nsq;
  int n_workers;
};

template <typename real_t> struct callBackData_t {
  real_t *Eq;
  real_t *nsq;
  real_t *qsum;
  uint64_t *qcount;
  real_t *eta;
  real_t *attrCoef;
  uint64_t *iter;
  uint64_t *maxIter;
};

// Callback, which is a CUDA host function that updates the progress meter
// and calculates Eq
template <typename real_t>
void CUDART_CB Eq_callback(void *data) {
  callBackData_t<real_t> *tmp = (callBackData_t<real_t> *)(data);
  real_t* Eq = tmp->Eq;
  real_t* nsq = tmp->nsq;
  real_t* qsum = tmp->qsum;
  uint64_t* qcount = tmp->qcount;
  *Eq = (*Eq * *nsq + *qsum) / (*nsq + *qcount);

  real_t* eta = tmp->eta;
  uint64_t* iter = tmp->iter;
  uint64_t* maxIter = tmp->maxIter;
  update_progress(*iter, *maxIter, *eta, *Eq);
}

// This is the class that does all the work
template <typename real_t> class SCEDeviceMemory {
public:
  SCEDeviceMemory(const std::vector<real_t> &Y, const std::vector<uint64_t> &I,
                  const std::vector<uint64_t> &J, const std::vector<double> &P,
                  const std::vector<real_t> &weights, int n_workers,
                  const int device_id, const unsigned int seed)
      : n_workers_(n_workers), nn_(weights.size()),
        ne_(P.size()), nsq_(static_cast<real_t>(nn_) * (nn_ - 1)),
        progress_callback_fn_(Eq_callback<real_t>),
        rng_state_(load_rng<real_t>(n_workers, seed)), Y_(Y), I_(I),
        J_(J), Eq_host_(1.0), Eq_device_(1.0),
        qsum_(n_workers), qsum_total_host_(0.0), qsum_total_device_(0.0),
        qcount_(n_workers), qcount_total_host_(0), qcount_total_device_(0) {
    // Initialise CUDA
    CUDA_CALL(hipSetDevice(device_id));

    // Initialise tmp space for reductions on qsum and qcount
    hipcub::DeviceReduce::Sum(qsum_tmp_storage_.data(), qsum_tmp_storage_bytes_,
                           qsum_.data(), qsum_total_device_.data(), qsum_.size());
    qsum_tmp_storage_.set_size(qsum_tmp_storage_bytes_);
    hipcub::DeviceReduce::Sum(qcount_tmp_storage_.data(),
                           qcount_tmp_storage_bytes_, qcount_.data(),
                           qcount_total_device_.data(), qcount_.size());
    qcount_tmp_storage_.set_size(qcount_tmp_storage_bytes_);

    // Set up discrete RNG tables
    node_table_ = set_device_table(weights);
    edge_table_ = set_device_table(P);
  }

  void runSCE(uint64_t maxIter, const int block_size,
      const int n_workers, const uint64_t nRepuSamp, real_t eta0,
      const bool bInit) {
    uint64_t iter = 0;
    real_t eta = eta0;
    real_t attrCoef = bInit ? 8 : 2;
    kernel_ptrs<real_t> device_ptrs = get_device_ptrs();

    // Set up a single iteration on a CUDA graph
    const size_t block_count = (n_workers_ + block_size - 1) / block_size;
    cuda_graph graph;
    hip_stream capture_stream, graph_stream;

    // Set up pointers used for kernel parameters in graph
    progress_callback_params_.Eq = &Eq_host_;
    progress_callback_params_.nsq = &nsq_;
    progress_callback_params_.qsum = &qsum_total_host_;
    progress_callback_params_.qcount = &qcount_total_host_;
    progress_callback_params_.eta = &eta;
    progress_callback_params_.attrCoef = &attrCoef;
    progress_callback_params_.iter = &iter;
    progress_callback_params_.maxIter = &maxIter;

    // SCE updates kernel with workers, then updates Eq
    // Start capture
    capture_stream.capture_start();
    wtsneUpdateYKernel<real_t><<<block_count, block_size, 0, capture_stream.stream()>>>(
        device_ptrs.rng, get_node_table(), get_edge_table(), device_ptrs.Y, device_ptrs.I, device_ptrs.J,
        device_ptrs.Eq, device_ptrs.qsum, device_ptrs.qcount, device_ptrs.nn,
        device_ptrs.ne, progress_callback_params_.eta, nRepuSamp, device_ptrs.nsq, progress_callback_params_.attrCoef, device_ptrs.n_workers);

    hipcub::DeviceReduce::Sum(qsum_tmp_storage_.data(), qsum_tmp_storage_bytes_,
                           qsum_.data(), qsum_total_device_.data(), qsum_.size(), capture_stream.stream());
    hipcub::DeviceReduce::Sum(qcount_tmp_storage_.data(),
                           qcount_tmp_storage_bytes_, qcount_.data(),
                           qcount_total_device_.data(), qcount_.size(), capture_stream.stream());
    qsum_total_device_.get_value_async(&qsum_total_host_, capture_stream.stream());
    qcount_total_device_.get_value_async(&qcount_total_host_, capture_stream.stream());

    capture_stream.add_host_fn(progress_callback_fn_, (void*)&progress_callback_params_);
    Eq_device_.set_value_async(&Eq_host_, capture_stream.stream());

    capture_stream.capture_end(graph.graph());
    // End capture

    // Main SCE loop - run captured graph
    for (iter = 0; iter < maxIter; iter++) {
      real_t new_eta = eta0 * (1 - static_cast<real_t>(iter) / (maxIter - 1));
      eta = MAX(new_eta, eta0 * 1e-4);
      attrCoef = (bInit && iter < maxIter / 10) ? 8 : 2;
      graph.launch(graph_stream.stream());
    }

    graph_stream.sync();
    std::cerr << std::endl << "Optimizing done" << std::endl;
  }

  std::vector<real_t> get_embedding_result() {
    std::vector<real_t> Y_host(Y_.size());
    Y_.get_array(Y_host);
    return Y_host;
  }

private:
  template <typename T>
  discrete_table_device<real_t> set_device_table(const std::vector<T>& probs) {
    discrete_table<real_t, T> table(probs);
    discrete_table_device<real_t> dev_table = { .F = table.F_table(),
                                         .A = table.A_table() };
    return dev_table;
  }

  discrete_table_ptrs<real_t> get_node_table() {
    discrete_table_ptrs<real_t> device_node_table = {.K = node_table_.F.size(),
                                                  .F = node_table_.F.data(),
                                                  .A = node_table_.A.data()};
    return device_node_table;
  }

  discrete_table_ptrs<real_t> get_edge_table() {
    discrete_table_ptrs<real_t> device_edge_table = {.K = edge_table_.F.size(),
                                                  .F = edge_table_.F.data(),
                                                  .A = edge_table_.A.data()};
    return device_edge_table;
  }

  kernel_ptrs<real_t> get_device_ptrs() {
    kernel_ptrs<real_t> device_ptrs = {.rng = rng_state_.data(),
                                       .Y = Y_.data(),
                                       .I = I_.data(),
                                       .J = J_.data(),
                                       .Eq = Eq_device_.data(),
                                       .qsum = qsum_.data(),
                                       .qcount = qcount_.data(),
                                       .nn = nn_,
                                       .ne = ne_,
                                       .nsq = nsq_,
                                       .n_workers = n_workers_};
    return device_ptrs;
  }

  // delete move and copy to avoid accidentally using them
  SCEDeviceMemory(const SCEDeviceMemory &) = delete;
  SCEDeviceMemory(SCEDeviceMemory &&) = delete;

  int n_workers_;
  uint64_t nn_;
  uint64_t ne_;
  real_t nsq_;

  hipHostFn_t progress_callback_fn_;
  callBackData_t<real_t> progress_callback_params_;

  // Uniform draw tables
  device_array<uint32_t> rng_state_;
  discrete_table_device<real_t> node_table_;
  discrete_table_device<real_t> edge_table_;

  // Embedding
  device_array<real_t> Y_;
  // Sparse distance indexes
  device_array<uint64_t> I_;
  device_array<uint64_t> J_;

  // Algorithm progress
  real_t Eq_host_;
  device_value<real_t> Eq_device_;
  device_array<real_t> qsum_;
  real_t qsum_total_host_;
  device_value<real_t> qsum_total_device_;
  device_array<uint64_t> qcount_;
  uint64_t qcount_total_host_;
  device_value<uint64_t> qcount_total_device_;

  // cub space
  size_t qsum_tmp_storage_bytes_;
  size_t qcount_tmp_storage_bytes_;
  device_array<void> qsum_tmp_storage_;
  device_array<void> qcount_tmp_storage_;
};

/****************************
 * Main control function     *
 ****************************/
// These two templates are explicitly instantiated here as the instantiation
// in python_bindings.cpp is not seen by nvcc, leading to a unlinked function
// when imported
template std::vector<float>
wtsne_gpu<float>(const std::vector<uint64_t> &, const std::vector<uint64_t> &,
                 std::vector<float> &, std::vector<float> &, const float,
                 const uint64_t, const int, const int, const uint64_t,
                 const float, const bool, const int, const int, const unsigned int);
template std::vector<double>
wtsne_gpu<double>(const std::vector<uint64_t> &, const std::vector<uint64_t> &,
                  std::vector<double> &, std::vector<double> &, const double,
                  const uint64_t, const int, const int, const uint64_t,
                  const double, const bool, const int, const int, const unsigned int);

template <typename real_t>
std::vector<real_t>
wtsne_gpu(const std::vector<uint64_t> &I, const std::vector<uint64_t> &J,
          std::vector<real_t> &dists, std::vector<real_t> &weights,
          const real_t perplexity, const uint64_t maxIter, const int block_size,
          const int n_workers, const uint64_t nRepuSamp, const real_t eta0,
          const bool bInit, const int cpu_threads, const int device_id,
          const unsigned int seed) {
  // Check input
  std::vector<real_t> Y;
  std::vector<double> P;
  std::tie(Y, P) =
      wtsne_init<real_t>(I, J, dists, weights, perplexity, cpu_threads, seed);

  // This class sets up and manages all of the memory
  SCEDeviceMemory<real_t> embedding(Y, I, J, P, weights, n_workers, device_id, seed);
  embedding.runSCE(maxIter, block_size, n_workers, nRepuSamp, eta0, bInit);
  return embedding.get_embedding_result();
}
